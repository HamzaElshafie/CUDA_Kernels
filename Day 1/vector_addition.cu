#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdlib>
#include <ctime>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N)
{
    // Element_id (i) = block_id * block_size + thread_id
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < N)
    {
        C[i] = A[i] + B[i]; // A[i] will translate to *(A + i)
    }
}

void vectorAddCPU(const float* A, const float* B, float* C, int N)
{
    for (int i = 0; i < N; ++i)
    {
        C[i] = A[i] + B[i];
    }
}

void initialiseVectors(float* A, float* B, int N)
{
    srand(static_cast<unsigned int>(time(0)));

    for (int i = 0; i < N; i++)
    {
        A[i] = static_cast<float>(rand()); // divide by RAND_MAX later if you want to normalise values
        B[i] = static_cast<float>(rand());
    }
}

template <typename Func>
double measureExecutionTime(Func func)
{
    auto start = std::chrono::high_resolution_clock::now();
    func();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> duration = end - start;
    return duration.count();
}

int main()
{
    int N = 1 << 20; // 1 million elements
    size_t size = N * sizeof(float); // Memory size needed to store the vectors for addition

    // Allocate memory on the host (CPU)
    float* A_host = (float*)malloc(size); // malloc return a void pointer
    float* B_host = (float*)malloc(size);
    float* C_host_cpu = (float*)malloc(size);
    float* C_host_gpu = (float*)malloc(size);

    initialiseVectors(A_host, B_host, N);

    // Measure CPU execution time for vector addition
    double cpu_time = measureExecutionTime([&]() 
    {
        vectorAddCPU(A_host, B_host, C_host_cpu, N);
    });

    std::cout << "CPU execution time: " << cpu_time << "ms" << '\n';

    
}

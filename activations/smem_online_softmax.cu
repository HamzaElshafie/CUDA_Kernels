#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <utils.h>

#define BLOCK_SIZE 1024

__global__ void smem_online_softmax(const float* __restrict__ A, float* __restrict__ C, int M, int N)
{
    extern __shared__ float smem[];

    // Each block will operate on a row
    int row = blockIdx.x;
    int t_idx = threadIdx.x;

    float local_max = -INFINITY;
    float local_norm = 0.0f;

    if (row < M)
    {
        // Phase 1: Calculate local norm and local max for each thread in block
        for (int i = t_idx; i < N; i += blockDim.x)
        {
            float x = A[row * N + i];
            if (x > local_max)
            {
                local_norm = local_norm * expf(local_max - x) + 1.0f; // 1.0f being the result of exp(x - new_max) if x was the new max
                local_max = x;
            }
            else
            {
                local_norm += expf(x - local_max);
            }
        }

        // Synchronise threads
        __syncthreads();
        // Store each thread local max in shared memory
        smem[t_idx] = local_max;
         __syncthreads();

    // Perform parallel reduction to obtain global max
    for (int stride = blockDim.x/2; stride > 0; stride/=2)
    {
        // Can't compare with out-of-bound thread
        if (t_idx < stride)
        {
            smem[t_idx] = fmaxf(smem[t_idx], smem[t_idx + stride]);
        }
        __syncthreads();
    }

    float global_max = smem[0];
    __syncthreads();

    // Rescale each thread local norm using global max and store in shared memory
    smem[t_idx] = local_norm * expf(local_max - global_max);
    __syncthreads();

    // Perform parallel reduction to obtain global normalisation factor
    for (int stride = blockDim.x/2; stride > 0; stride/=2)
    {
        if (t_idx < stride)
        {
            smem[t_idx] = smem[t_idx] + smem[t_idx + stride];
        }
        __syncthreads();
    }

    float global_norm = smem[0];
    __syncthreads(); // not needed but kept for good practice

    // Phase 2: Get softmax output
    for (int i = t_idx; i < N; i+=blockDim.x)
    {
        int index = row * N + i;
        C[index] = expf(A[index] - global_max) / global_norm;
    }
    }
}

void online_softmax_cpu(const float* A, float* C, int M, int N) {
    for (int row = 0; row < M; row++) {
        // Phase 1: Find the maximum value using the online algorithm
        float local_max = -INFINITY;
        float local_norm = 0.0f;
        
        for (int col = 0; col < N; col++) {
            float x = A[row * N + col];
            if (x > local_max) {
                local_norm = local_norm * expf(local_max - x) + 1.0f; // Same adjustment as CUDA
                local_max = x;
            } else {
                local_norm += expf(x - local_max);
            }
        }
        
        float global_max = local_max;
        float global_norm = local_norm;
        
        // Phase 2: Calculate softmax values
        for (int col = 0; col < N; col++) {
            C[row * N + col] = expf(A[row * N + col] - global_max) / global_norm;
        }
    }
}

int main()
{
    // Set matrix and block dimensions
    int num_rows = 1 << 10; // M
    int num_columns = 1 << 10; // N

    size_t size = (num_rows * num_columns) * sizeof(float);

    // Allocate host memory
    float* A_host = (float*)malloc(size);
    float* C_host_cpu = (float*)malloc(size);
    float* C_host_gpu = (float*)malloc(size);

    // Initialise matrix
    float* a_array[] {A_host};
    initialiseArrays(a_array, 1, num_rows * num_columns, -100.0f, 100.0f, 0);

    // Measure CPU execution time
    double cpu_time = measureExecutionTime([&](){
        online_softmax_cpu(A_host, C_host_cpu, num_columns, num_columns);
    });

    std::cout << "CPU execution time: " << cpu_time << "ms" << std::endl;

    // Allocate device memory
    float* A_device;
    float* C_device;

    CUDA_CHECK(hipMalloc((void**)&A_device, size));
    CUDA_CHECK(hipMalloc((void**)&C_device, size));

    // Copy data from host 
    CUDA_CHECK(hipMemcpy(A_device, A_host, size, hipMemcpyHostToDevice));

    // Define kernel configs
    int threads_per_block = 1024;
    int blocks_per_grid = num_rows;
    size_t smem_size = threads_per_block * sizeof(float);
    dim3 blockDim(threads_per_block);
    dim3 gridDim(blocks_per_grid);

    std::cout << "Grid configuration: " << std::endl;
    std::cout << "Grid dimension: " << gridDim.x << std::endl;
    std::cout << "Block dimension: " << blockDim.x << std::endl;

    // Measure GPU execution time
    float gpu_time = measureKernelTime([&](){
        smem_online_softmax<<<gridDim, blockDim, smem_size>>>(A_device, C_device, num_rows, num_columns);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    });

    std::cout << "GPU execution time: " << gpu_time << "ms" << std::endl;
    std::cout << "Speedup: " << cpu_time / gpu_time << "x" << std::endl;

    // Calculate throughput

    // Copy results to host
    CUDA_CHECK(hipMemcpy(C_host_gpu, C_device, size, hipMemcpyDeviceToHost));

    // Verify results
    bool results_match = compareResults(C_host_cpu, C_host_gpu, num_rows * num_columns, 1e-4, 1e-5);
    std::cout << (results_match? "Results match!" : "Results do not match!") << std::endl;

    // Free device memory
    CUDA_CHECK(hipFree(A_device));
    CUDA_CHECK(hipFree(C_device));

    // Free host memory
    free(A_host);
    free(C_host_cpu);
    free(C_host_gpu);

    return 0;
}

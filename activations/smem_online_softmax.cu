#include "hip/hip_runtime.h"
#include <iostream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cmath>

#include <utils.h>

#def BLOCK_SIZE 1024

__global__ void online_softmax(const float* __restrict__ A, float* __restrict__ C, int M, int N)
{
    __shared float smem[BLOCK_SIZE];

    // Each block will operate on a row
    int row = blockIdx.x;
    int t_idx = threadIdx.x;

    float local_max = -INFINITY;
    float local_norm = 0.0f;

    if (row < M)
    {
        // Phase 1: Calculate local norm and local max for each thread in block
        for (int i = t_idx; i < N; i += blockDim.x)
        {
            float x = A[row * N + i];
            if (x > local_max)
            {
                local_norm *= expf(local_max - x);
                local_max = x;
            }
            local_norm += expf(x - local_max);
        }
        // Synchronise threads
        __syncthreads();
        // Store each thread local max in shared memory
        smem[t_idx] = local_max;
         __syncthreads();

    // Perform paralell reduction to obtain global max
    for (int stride = blockDim.x/2; stride > 0; stride/=2)
    {
        if (t_idx < stride)
        {
            smem[t_idx] = fmaxf(smem[t_idx], smem[t_idx + stride]);
        }
        __syncthreads();
    }
    float global_max = smem[0];
    __syncthreads();

    // Perform paralell reduction to obtain global normalisation factor

    // Phase 2: Get softmax output

    }
}

int main()
{
    // Set matrix and block dimensions
    int num_rows = 1 << 10; // M
    int num_columns = 1 << 10; // N

    size_t size = (num_rows * num_columns) * sizeof(float);

    // Allocate host memory
    float* A_host = (float*)malloc(size);
    float* C_host_cpu = (float*)malloc(size);
    float* C_host_gpu = (float*)malloc(size);

    // Initialise matrix
    float* a_array[] {A_host};
    initialiseArrays(a_array, 1, num_rows * num_columns, -100.0f, 100.0f, 0);

    // Measure CPU execution time

    // Allocate device memory
    float* A_device;
    float* C_device;

    CUDA_CHECK(hipMalloc((void**)&A_device, size));
    CUDA_CHECK(hipMalloc((void**)&C_device, size));

    // Copy data from host 
    CUDA_CHECK(hipMemcpy(A_device, A_host, size, hipMemcpyHostToDevice));

    // Define kernel configs
    int threads_per_block = 256;
    int blocks_per_grid = (num_rows + threads_per_block - 1) / threads_per_block;
    dim3 blockDim(threads_per_block);
    dim3 gridDim(blocks_per_grid);

    std::cout << "Grid configuration: " << std::endl;
    std::cout << "Grid dimension: " << gridDim.x << std::endl;
    std::cout << "Block dimension: " << blockDim.x << std::endl;

    // Measure GPU execution time
    float gpu_time = measureKernelTime([&](){
        online_softmax<<<gridDim, blockDim>>>(A_device, C_device, num_rows, num_columns);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    });

    std::cout << "GPU execution time: " << gpu_time << "ms" << std::endl;

    // Calculate throughput

    // Copy results to host
    CUDA_CHECK(hipMemcpy(C_host_gpu, C_device, size, hipMemcpyDeviceToHost));

    // Verify results

    // Free device memory
    CUDA_CHECK(hipFree(A_device));
    CUDA_CHECK(hipFree(C_device));

    // Free host memory
    free(A_host);
    free(C_host_cpu);
    free(C_host_gpu);

    return 0;
}
